#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

float* Acpu;
float* Agpu;
float* Bcpu;
float* Bgpu;
float* Ccpu;
float* Ccpu2;
float* Cgpu;

//Cuda error checking - non mandatory
void cudaCheckError() {
    hipError_t e = hipGetLastError();
    if(e != hipSuccess) {
        printf("Cuda failure %s:%d: '%s' : %d\n", __FILE__, __LINE__, hipGetErrorString(e), e);
        exit(0);
    }
}

float* cpu_matrix_allocation(int rows, int cols){
	float* M = (float*) malloc(cols * rows * sizeof(float));
	return M;
}

float* gpu_matrix_allocation(float* M, int rows, int cols){
	hipMalloc((void**) &(M), rows * cols * sizeof(float));
	return M;
}

void init_matrix(float* A, int rows, int cols){
	for(int i = 0; i < rows; ++i){
		for(int j = 0; j < cols; ++j){
			A[i*cols + j]=(i+1);
		}
	}
}

void print_matrix(float* A, int rows, int cols){
	for(int i = 0; i < rows; ++i){
		for(int j = 0; j < cols; ++j){
			printf("%f ", A[i*cols+j]);
		}
		printf("\n");
	}
	printf("\n");
}

void matmul_cpu(float* A, float* B, float* C, int rowsa, int colsa, int rowsb, int colsb) {

	if(colsa != rowsb){
    	printf("Error in matrix dimensions\n");
    	exit(0);
    }

	for(int row = 0; row < rowsa; ++row){
		for(int col = 0; col < colsb; ++col){
			float sum = 0;
	        for (int i = 0; i < colsa; i++) {
	            sum += A[row * colsa + i] * B[i * colsb + col];
	        }
	        C[row * colsb + col] = sum;
		}
	}
    
}

__global__ void matmul_kernel(float* A, float* B, float* C, int rowsa, int colsa, int rowsb, int colsb) {

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float sum = 0;

    if(colsa != rowsb){
    	printf("Error in matrix dimensions\n");
    	
    }
    else{
    	if (row < rowsa && col < colsb) {
	        // each thread computes one element of the block sub-matrix
	        for (int i = 0; i < colsa; i++) {
	            sum += A[row * colsa + i] * B[i * colsb + col];
	        }
	        C[row * colsb + col] = sum;
	    }
	    
    }
}

void check_equal(float* C, float* C2, int rows, int cols){
	for(int i = 0; i < rows*cols; ++i){
		if(C[i] != C2[i]){
			printf("Error in multiplication at row %d and column %d, data %f, %f\n", 
				i/cols ,i%cols, C[i], C2[i]);
			break;
		}
	}	
	printf("Assertion end\n");
}

void all(int rowsa, int colsa, int rowsb, int colsb, int thread_no_x, int thread_no_y){
	// Constants calculation
	int rowsc = rowsa; int colsc = colsb;
	int nbytesa = rowsa*colsa*sizeof(float);
	int nbytesb = rowsb*colsb*sizeof(float);
	int nbytesc = rowsc*colsc*sizeof(float);
	int block_no_x = (colsc + thread_no_x -1)/thread_no_x;//ceil(colsc/thread_no_x);
	int block_no_y = (rowsc + thread_no_y -1)/thread_no_y;//ceil(rowsc/thread_no_y);
	dim3 threadsPerBlock(thread_no_x, thread_no_y);
    dim3 blocksPerGrid(block_no_x, block_no_y);

	printf("Begin\n");

	//Allocation
	Acpu = cpu_matrix_allocation(rowsa, colsa);
	Bcpu = cpu_matrix_allocation(rowsb, colsb);
	Ccpu = cpu_matrix_allocation(rowsc, colsc);
	Ccpu2 = cpu_matrix_allocation(rowsc, colsc);
	printf("Allocated cpu\n");
	Agpu = gpu_matrix_allocation(Agpu, rowsa, colsa);
	Bgpu = gpu_matrix_allocation(Bgpu, rowsb, colsb);
	Cgpu = gpu_matrix_allocation(Cgpu, rowsc, colsc);
	printf("Allocated gpu\n");

	// Initializes matrices
	init_matrix(Acpu, rowsa, colsa);
	init_matrix(Bcpu, rowsb, colsb);
	hipMemcpy(Agpu, Acpu, nbytesa, hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, Bcpu, nbytesb, hipMemcpyHostToDevice);
	printf("Initialized matrices\n");

	// Multiply cpu
	clock_t start = clock();
	matmul_cpu(Acpu, Bcpu, Ccpu, rowsa, colsa, rowsb, colsb);
	clock_t time_cpu = clock()-start;
	printf("Multiplied cpu\n");

	// Multiply gpu
	start = clock();
    matmul_kernel<<<blocksPerGrid, threadsPerBlock>>>(Agpu, Bgpu, Cgpu, rowsa, colsa, rowsb, colsb);
    hipDeviceSynchronize();
    clock_t time_gpu = clock()-start;
    printf("Multiplied gpu\n");

    // rowsc, colsc, b_x, b_y, gpu time, cpu time
	printf("%d,%d,%d,%d,%f,%f\n", 
		rowsc, colsc, block_no_x,block_no_y,(double)time_gpu/CLOCKS_PER_SEC,(double)time_cpu/CLOCKS_PER_SEC);
    hipMemcpy(Ccpu2, Cgpu, nbytesc, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    //print_matrix(Ccpu2, rowsc, colsc);
    //print_matrix(Ccpu, rowsc, colsc);
    check_equal(Ccpu, Ccpu2, rowsc, colsc);
}

int main(int argc, char** argv){

	// Variables
	int rowsa = 1000; int colsa = 445;
	int rowsb = 445; int colsb = 1000;
	int thread_no_x = 100; int thread_no_y = 10; // tx * ty < 1024!

	all(rowsa, colsa, rowsb, colsb, thread_no_x, thread_no_y);

}