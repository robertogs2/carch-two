#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

float* Ycpu;
float* Ygpu;
float* Xcpu;
float* Xgpu;

//Cuda error checking - non mandatory
void cudaCheckError() {
    hipError_t e = hipGetLastError();
    if(e != hipSuccess) {
        printf("Cuda failure %s:%d: '%s' : %d\n", __FILE__, __LINE__, hipGetErrorString(e), e);
        exit(0);
    }
}

float* cpu_allocation(int n){
	return (float*) malloc(n * sizeof(float));
}

float* gpu_allocation(float* vector, int n){
	hipMalloc((void**) &(vector), n * sizeof(float));
	return vector;
}

void init_vector(float* vector, int n){
	for(int i = 0; i < n; ++i){
		vector[i]=(i+1);
	}
}

void print_vec(float* vector, int n){
	for(int i = 0; i < n; ++i){
		printf("%f ", vector[i]);
	}
	printf("\n");
}

void saxpy_cpu(float* y, float* x, float a, int n) {

	for(int i = 0; i < n; ++i){
		y[i] = a*x[i]+y[i];
	}
    
}

__global__ void saxpy_kernel(float* y, float* x, float a, int n) {

    int index = blockIdx.x*blockDim.x+threadIdx.x;

    if(index < n) y[index] = a*x[index]+y[index];
}

void check_equal(float* v1, float* v2, int n){
	for(int i = 0; i < n; ++i){
		if(v1[i] != v2[i]){
			printf("Error at index %d", i);
			break;
		}
	}	
}

void all(int n, int thread_no){
	// Constants calculation
	int nbytes = n*sizeof(float);
	int block_no = (n + thread_no -1)/thread_no;//ceil(colsc/thread_no);
	dim3 threadsPerBlock(thread_no, 1);
    dim3 blocksPerGrid(block_no, 1);

	//Allocation
	Ycpu = cpu_allocation(n);
	Xcpu = cpu_allocation(n);
	Xgpu = gpu_allocation(Xgpu, n);
	Ygpu = gpu_allocation(Ygpu, n);

	// Initializes matrices
	init_vector(Ycpu, n);
	init_vector(Xcpu, n);
	hipMemcpy(Ygpu, Ycpu, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(Xgpu, Xcpu, nbytes, hipMemcpyHostToDevice);

	// Saxpy cpu
	clock_t start = clock();
	saxpy_cpu(Ycpu, Xcpu, 2, n);
	clock_t time_cpu = clock()-start;

	// Saxpy gpu
	start = clock();
    saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(Ygpu, Xgpu, 2, n);
    hipDeviceSynchronize();
    clock_t time_gpu = clock()-start;
  

    // n, blocks, gpu time, cpu time
	printf("%d,%d,%f,%f\n", 
		n, block_no,(double)time_gpu/CLOCKS_PER_SEC,(double)time_cpu/CLOCKS_PER_SEC);
    //hipMemcpy(Ccpu2, Cgpu, nbytesc, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    //print_vector(Ccpu2, rowsc, colsc);
    //print_vector(Ccpu, rowsc, colsc);
    //check_equal(Xcpu, Ccpu2, rowsc, colsc);
}

int main(int argc, char** argv){

	int thread_no = 100;
	// Variables
	for(int size = 4; size <= 10000; size+=100){
		all(size, thread_no);
	}

}
